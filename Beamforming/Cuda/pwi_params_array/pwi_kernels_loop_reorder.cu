
#include <hip/hip_runtime.h>
#define MAX_FIR_SIZE 64
#define FLT_EPSILON 1.1920929e-7f

enum FloatParams {
    FS,
    C1,
    PITCH,
    F1,
    F2,
    BFD,
    X_STEP,
    Z_STEP,
    X0_ROI,
    Z0_ROI,
    T_START,
    X_0,
    FLOAT_PARAMS_COUNT // Número total de parámetros float
};

enum IntParams {
    TAPS,
    N_ELEMENTOS,
    N_ANGLES,
    NX,
    NZ,
    N_SAMPLES,
    INT_PARAMS_COUNT // Número total de parámetros int
};


__device__ void shift_izq(short *x, int taps) {
    for (int j = 0; j < taps; j++) {
        x[j] = x[j + 1];
    }
    x[taps] = 0;
}


__device__ float multisum(short *x, float *coef, int taps) {
    float q = 0;
    for (int j = 0; j < (taps + 1); j++) {
        q += coef[taps - j] * x[j];
        // Limitar el valor acumulado dentro del rango de un short
    }
    q = fmaxf(fminf(q, 32767.0f), -32768.0f);
    return q;
}

__device__ void compute_sample_index(float *x_rx, float *xf, float *zf, float *c1, float *bfd,
                                     float *fs, int *ns, float *t1, float *t2, float *t,
                                     unsigned int *k, float *ap_dyn) {
    *t2 = hypotf(*x_rx - *xf, *zf) / *c1;
    *ap_dyn = fabsf(*x_rx - *xf) / *zf < *bfd;
    *ap_dyn = *zf/(fabsf(*x_rx - *xf) + FLT_EPSILON) > *bfd ;  // Apodización dinámica
    *t = *t1 + *t2;
    *t = *t * (*t > 0 ? 1 : 0);  /* First sample must be 0 !!! */
    *k = min((unsigned int)floorf(*t * (*fs)), *ns - 2); /* resto 2 para evitar que k+1 = ns */
}



extern "C" __global__ void filt_kernel(const int *int_params, const short *datain, const float *coef_g, short *dataout)
    {

    // Calcular índices globales del hilo
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Índice de thread

    int nel = int_params[N_ELEMENTOS];
    int ns = int_params[N_SAMPLES];
    int taps = int_params[TAPS];

    // Verificar límites
    if (tid >= nel * ns) return;

    //extern __shared__ float coef[];
    float coef[MAX_FIR_SIZE] ;
    short x[MAX_FIR_SIZE];

    // Índice de primer sample del ascan en dataout
    int i = ns * tid;

    /* copiar coeficientes del filtro en memoria privada */
    unsigned short l0 = taps/2 ;
    unsigned short lmax = taps + 1 ;
    for (unsigned short l=0; l < lmax; l++) {
        coef[l] = coef_g[l] ;
    }

    // Calcular transitorio
    for (unsigned short l = 0; l <= l0; l++) {
        x[l + l0] = datain[i + l];
    }

    // Calcular primera muestra de salida
    dataout[i] = (short)rintf(multisum(x, coef, taps));

    // Continuar hasta traer la última muestra del A-scan
    lmax = ns - l0;
    for (unsigned short l = 1; l < lmax; l++) {
        shift_izq(x, taps);
        x[taps] = datain[i + l + l0];
        dataout[i + l] = (short)rintf(multisum(x, coef, taps));
    }

    for (int l = lmax; l < ns; l++) {
        shift_izq(x, taps);
        dataout[i + l] = (short)rintf(multisum(x, coef, taps));
    }

    /* forzar esto para el tema de los bordes en el beamforming */
    dataout[i] = 0 ;
    dataout[i + ns - 2] = 0 ;
    dataout[i + ns - 1] = 0 ;
}

// cambiar el orden del loop, INCOMPLETO!!!
extern "C" __global__ void pwi_1pix_per_thread(
                               const int *int_params,
                               const float *float_params,
                               const float *angles,
                               const short *matrix,
                               const short *matrix_imag,
                               float *img,
                               float *img_imag,
                               float *cohe) {

    unsigned short iz = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned short ix = blockIdx.y * blockDim.y + threadIdx.y;

    // Obtener los parámetros enteros y flotantes
    // int_params
    int nel = int_params[N_ELEMENTOS];
    int nang = int_params[N_ANGLES];
    int ns = int_params[N_SAMPLES];
    int nx = int_params[NX];
    int nz = int_params[NZ];
    if (iz >= nz || ix >= nx) return;  // Verificar límites de los índices

    // float params
    float fs = float_params[FS];
    float c1 = float_params[C1];
    float pitch = float_params[PITCH];
    float x0_roi = float_params[X0_ROI];
    float z0_roi = float_params[Z0_ROI];
    float x_step = float_params[X_STEP];
    float z_step = float_params[Z_STEP];
    float t_start = float_params[T_START];
    float x0 = float_params[X_0];
    float bfd = float_params[BFD];

    float xf = x0_roi + x_step * ix;
    float zf = z0_roi + z_step * iz;  // Z POSITIVE DOWNWARDS
    float x_rx = -x0, wave_source;
    float t1, t2;
    float t, dt, temp, theta, ap_dyn;
    unsigned int k, k0 = 0;
    float a, b, q = 0, q_imag = 0, w = 0, w_imag = 0;

    unsigned short f_idx = iz * nx + ix;

    for (unsigned short e = 0; e < nel; e++) {
        k0 = e * ns;  // Índice base para el A-scan
        x_rx += pitch;  // Incrementar x_rx para cada elemento
        t2 = hypotf(x_rx - xf, zf) / c1;
        ap_dyn = zf/(fabsf(x_rx - xf) + FLT_EPSILON) > bfd ;  // Apodización dinámica

        for (unsigned short i = 0; i < nang; i++) {

            theta = angles[i];
            wave_source = x0 * (theta < 0 ? 1 : -1);
            t1 = ((xf - wave_source) * sinf(theta) + zf * cosf(theta)) / c1 - t_start;
            t = t1 + t2;
            dt = t * fs - k;

            temp = (float)matrix[k0 + k];
            a = ((float)matrix[k0 + k + 1] - temp) * dt + temp;
            q += a * ap_dyn;

            temp = (float)matrix_imag[k0 + k];
            b = ((float)matrix_imag[k0 + k + 1] - temp) * dt + temp;
            q_imag += b * ap_dyn;

            temp = hypotf(a, b) + FLT_EPSILON;  /* módulo del "fasor" */
            /* se suman las componentes de los fasores para cada A-scan */
            w += a / temp;
            w_imag += b / temp;

            k0 += ns * nel; // saltar a la siguiente onda
        }
    }
    img[f_idx] = q ;
    img_imag[f_idx] = q_imag ;
    cohe[f_idx] = hypotf(w, w_imag) ;
}
