#include "hip/hip_runtime.h"
extern "C" __global__ void filt_batch_kernel(const int *int_params, const short *datain, const float *coef_g, short *dataout)
    {

    // Calcular índices globales del hilo
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Índice de thread

    int nang = int_params[N_ANGLES];
    int nel = int_params[N_ELEMENTOS];
    int ns = int_params[N_SAMPLES];
    int taps = int_params[TAPS];
    int n_batch = int_params[N_BATCH];
    int batch_stride = ns * nel * nang; // stride para indexar los batches
    int i ;

    // Verificar límites
    if (tid >= nel * ns) return;

    //extern __shared__ float coef[];
    float coef[MAX_FIR_SIZE] ;
    short x[MAX_FIR_SIZE];

    /* copiar coeficientes del filtro en memoria privada */
    unsigned short l0 = taps/2 ;
    unsigned short lmax = taps + 1 ;
    for (unsigned short l=0; l < lmax; l++) {
        coef[l] = coef_g[l] ;
        }

    lmax = ns - l0;
    for (unsigned short bch = 0; bch < n_batch; bch++) {

        i = ns * tid + bch * batch_stride; //índice para el siguiente batch

        // Calcular transitorio
        for (unsigned short l = 0; l <= l0; l++) {
            // NOTA: segun lo que haya en x, puede habr un problemita en el transitorio, porque la
            // primera mitad de x debería tener valore nulos, y puede que no sea así. Se podría inicilizar
            // para evitar esto
            x[l + l0] = datain[i + l];
        }

        // Calcular primera muestra de salida
        dataout[i] = (short)rintf(multisum(x, coef, taps));

        // Continuar hasta traer la última muestra del A-scan
        for (unsigned short l = 1; l < lmax; l++) {
            shift_izq(x, taps);
            x[taps] = datain[i + l + l0];
            dataout[i + l] = (short)rintf(multisum(x, coef, taps));
        }

        for (int l = lmax; l < ns; l++) {
            shift_izq(x, taps);
            dataout[i + l] = (short)rintf(multisum(x, coef, taps));
        }

        /* forzar esto para el tema de los bordes en el beamforming */
        dataout[i] = 0 ;
        dataout[i + ns - 2] = 0 ;
        dataout[i + ns - 1] = 0 ;
    }
}


extern "C" __global__ void pwi_batch_1pix_per_thread(
                               const int *int_params,
                               const float *float_params,
                               const float *angles,
                               const short *matrix,
                               const short *matrix_imag,
                               float *img,
                               float *img_imag) {

    unsigned short iz = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned short ix = blockIdx.y * blockDim.y + threadIdx.y;

    // Obtener los parámetros enteros y flotantes
    // int_params
    int n_batch = int_params[N_BATCH];
    int nel = int_params[N_ELEMENTOS];
    int nang = int_params[N_ANGLES];
    int ns = int_params[N_SAMPLES];
    int nx = int_params[NX];
    int nz = int_params[NZ];
    if (iz >= nz || ix >= nx) return;  // Verificar límites de los índices

    // float params
    float fs = float_params[FS];
    float c1 = float_params[C1];
    float pitch = float_params[PITCH];
    float x0_roi = float_params[X0_ROI];
    float z0_roi = float_params[Z0_ROI];
    float x_step = float_params[X_STEP];
    float z_step = float_params[Z_STEP];
    float t_start = float_params[T_START];
    float x0 = float_params[X_0];
    float bfd = float_params[BFD];

    float xf = x0_roi + x_step * ix;
    float zf = z0_roi + z_step * iz;  // Z POSITIVE DOWNWARDS
    float x_rx, wave_source;
    float t1, t2;
    float t, dt, temp, theta, ap_dyn;
    float a, b; // para amacenar las samples localmente

    // variables para indexar
    unsigned int k, k0 = 0;
    unsigned int m_idx, batch_offset; // para indexar las matrix
    unsigned int batch_stride = ns * nel * nang, img_stride = nx * nz ;
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x; // Para indexar la memoria compartida
    unsigned int q_stride = blockDim.x * blockDim.y; // stride para indexar la memoria compartida
    unsigned int q_idx ; // valor inicial para indexar la memoria compartida

    unsigned int f_idx = iz * nx + ix; // para indexar la imagen

    extern __shared__ float q[]; // Variable compartida para almacenar resultados parciales, el tamaño
    // es (nro de pixeles por block) * n_batch * 2 (real e imaginaria)
    for (unsigned int i = 0; i < n_batch * 2; i++) {
        q[tid + i * q_stride] = 0.0f;  // Inicializar memoria compartida
    }
    __syncthreads();

    for (unsigned short i = 0; i < nang; i++) {
        theta = angles[i];
        wave_source = x0 * (theta < 0 ? 1 : -1);
        t1 = ((xf - wave_source) * sinf(theta) + zf * cosf(theta)) / c1 - t_start;
        x_rx = -x0;  // Inicializar x_rx para el primer elemento

        for (unsigned short e = 0; e < nel; e++) {
            x_rx += pitch;  // Incrementar x_rx para cada elemento
            compute_sample_index(&x_rx, &xf, &zf, &c1, &bfd, &fs, &ns, &t1, &t2, &t, &k, &ap_dyn);
            dt = t * fs - k;

            batch_offset = 0 ;
            q_idx = tid ;
            for (unsigned short bch = 0; bch < n_batch; bch++) {
                m_idx = batch_offset + k0 + k;
                temp = (float)matrix[m_idx];
                a = ((float)matrix[m_idx + 1] - temp) * dt + temp;
                __syncthreads();
                q[q_idx] += a * ap_dyn;

                temp = (float)matrix_imag[m_idx];
                b = ((float)matrix_imag[m_idx + 1] - temp) * dt + temp;
                q_idx += q_stride; // Incrementar el índice para la memoria compartida
                __syncthreads();
                q[q_idx] += b * ap_dyn;
                batch_offset += batch_stride ;
                q_idx += q_stride; // Incrementar el índice para la memoria compartida
            }
            k0 += ns;
        }
    }

    q_idx = tid; // Reiniciar el índice para la memoria compartida
    for (unsigned short bch = 0; bch < n_batch; bch++) {
         __syncthreads();
        img[f_idx] = q[q_idx] ;
        __syncthreads();
        q_idx += q_stride; // Incrementar el índice para la memoria compartida
        __syncthreads();
        img_imag[f_idx] = q[q_idx];
        f_idx += img_stride; // Incrementar el índice para la imagen
        q_idx += q_stride; // Incrementar el índice para la memoria compartida
        }
}